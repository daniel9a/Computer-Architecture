
#include <hip/hip_runtime.h>
#include <chrono>
#include <iostream>
#include <math.h>

typedef std::chrono::high_resolution_clock Clock;
#define NUM_THREADS_IN_BLOCK 256


__global__
//runtime GPU 195.58us
//runtime CPU 3015 microseconds 
void daxpyGPU(int arraySize, float *a, float *b, float *c, float *result)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < arraySize; i+=stride) {
        result[i] = a[i] * b[i] + c[i];
    }
  // Insert your code here.
} // daxpyGPU()

//runtime 3015 microseconds 
void daxpyCPU(int arraySize, float *a, float *x, float *y, float *result)
{
  for(int index = 0; index < arraySize; index++)
  {
    result[index] = a[index] * x[index] + y[index];
  }
} // daxpyCPU()

int main(void)
{
  int arraySize = 1 << 20;
  float *a, *x, *y;
  float *cpuResult, *gpuResult;

  cpuResult = new float[arraySize];

  // Allocate unified memory, accessible from CPU or GPU.
  hipMallocManaged(&a, arraySize * sizeof(float));
  hipMallocManaged(&x, arraySize * sizeof(float));
  hipMallocManaged(&y, arraySize * sizeof(float));
  hipMallocManaged(&gpuResult, arraySize * sizeof(float));

  // Initialize arrays on the host.
  for(int index = 0; index < arraySize; index++)
  {
    a[index] = 5.0f;
    x[index] = 10.0f;
    y[index] = 20.0f;
  }

  int blockSize = NUM_THREADS_IN_BLOCK;
  int numBlocks = (arraySize + blockSize - 1) / blockSize;

  auto start = Clock::now();
  daxpyCPU(arraySize, a, x, y, cpuResult);
  auto end = Clock::now();

  daxpyGPU<<<numBlocks, blockSize>>>(arraySize, a, x, y, gpuResult);

  // Wait for GPU to finish before accessing values on the host.
  hipDeviceSynchronize();

  // Check for errors. All values should be 70.0f.
  float maxError = 0.0f;

  for (int index = 0; index < arraySize; index++)
  {
    maxError = fmax(maxError, fabs(cpuResult[index] - gpuResult[index]));
  }

  std::cout << "Max error: " << maxError << std::endl
            << "CPU time: "
            << std::chrono::duration_cast<std::chrono::microseconds>(end - start).count()
            << " microseconds." << std::endl;

  // Free memory.
  hipFree(a);
  hipFree(x);
  hipFree(y);
  hipFree(gpuResult);
  delete(cpuResult);

  return 0;
} // main()
